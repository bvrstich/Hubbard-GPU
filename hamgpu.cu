#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include "hamgpu.h"

// number of threads in a block (must be multiple of 32)
#define NUMTHREADS 256

#define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }


GPUHamiltonian::GPUHamiltonian(int Ns, int Nu, int Nd, double J, double U)
    : SparseHamiltonian(Ns,Nu,Nd,J,U)
{
}

GPUHamiltonian::~GPUHamiltonian()
{
}

__global__ void gpu_mvprod(double *x, double *y, double alpha, int NumUp, int NumDown, int dim, double *Umat, double *Down_data,unsigned int *Down_ind, int size_Down, double *Up_data, unsigned int *Up_ind, int size_Up, int rows_shared)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < dim)
    {
	double result = Umat[index] * x[index];

	int sv = index / NumDown; //__fdividef(index,NumDown);
	int id = index % NumDown; // index - sv*NumDown;

	extern __shared__ double shared[];

	if(threadIdx.x <= rows_shared)
	    for(int i=0;i<size_Up;i++)
		shared[threadIdx.x*size_Up+i] = Up_data[(blockDim.x * blockIdx.x)/NumDown + threadIdx.x + i*NumUp];

	__syncthreads();

	for(int i=0;i<size_Up;i++)
//	    result += Up_data[sv+i*NumUp] * x[id + NumDown*Up_ind[sv+i*NumUp]];
	    result += shared[(sv-(blockDim.x * blockIdx.x)/NumDown)*size_Up+i] * x[id + NumDown*Up_ind[sv+i*NumUp]];

	for(int i=0;i<size_Down;i++)
	    result += Down_data[id+i*NumDown] * x[sv*NumDown + Down_ind[id+i*NumDown]];

	y[index] = alpha * y[index] + result;
    }
}

void GPUHamiltonian::mvprod(double *x, double *y, double alpha)
{
    int NumUp = baseUp.size();
    int NumDown = baseDown.size();
    int dim = NumUp*NumDown;
    int numblocks = ceil(dim*1.0/NUMTHREADS);
    int rows_shared = ceil(NUMTHREADS*1.0/NumDown);
    size_t sharedmem = size_Up * (rows_shared+1) * sizeof(double);

    hipGetLastError();
    gpu_mvprod<<<numblocks,NUMTHREADS,sharedmem>>>(x,y,alpha,NumUp,NumDown,dim,Umat_gpu,Down_data_gpu,Down_ind_gpu,size_Down,Up_data_gpu,Up_ind_gpu,size_Up,rows_shared);
    CUDA_SAFE_CALL(hipGetLastError());
}

double GPUHamiltonian::LanczosDiagonalize(int m)
{
    int device;
    hipGetDevice( &device );

    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, device );

    int NumUp = baseUp.size();
    int NumDown = baseDown.size();

    size_t neededmem = getDim()*sizeof(double) +
	2*NumUp*size_Up*sizeof(double) +
	2*NumDown*size_Down*sizeof(double) +
	2*dim*sizeof(double);

    if(neededmem > prop.totalGlobalMem)
    {
	std::cerr << "Houston, we have a memory problem!" << std::endl;
	return 0;
    }

    if( ceil(dim*1.0/NUMTHREADS) > prop.maxGridSize[0] )
    {
	std::cerr << "Houston, we have a grid size problem!" << std::endl;
	return 0;
    }

    if( (ceil(NUMTHREADS/NumDown) + 1) * size_Up * sizeof(double) > prop.sharedMemPerBlock )
    {
	std::cerr << "Houston, we have a shared memory size problem!" << std::endl;
	return 0;
    }

    // alloc Umat and copy to gpu
    double *Umat = Umatrix();
    CUDA_SAFE_CALL(hipMalloc(&Umat_gpu, getDim()*sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpy(Umat_gpu,Umat,getDim()*sizeof(double),hipMemcpyHostToDevice));

    delete [] Umat;


    CUDA_SAFE_CALL(hipMalloc(&Up_data_gpu,NumUp*size_Up*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&Up_ind_gpu,NumUp*size_Up*sizeof(double)));

    CUDA_SAFE_CALL(hipMemcpy(Up_data_gpu,Up_data,NumUp*size_Up*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(Up_ind_gpu,Up_ind,NumUp*size_Up*sizeof(double),hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMalloc(&Down_data_gpu,NumDown*size_Down*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&Down_ind_gpu,NumDown*size_Down*sizeof(double)));

    CUDA_SAFE_CALL(hipMemcpy(Down_data_gpu,Down_data,NumDown*size_Down*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(Down_ind_gpu,Down_ind,NumDown*size_Down*sizeof(double),hipMemcpyHostToDevice));

    double *a = new double[m];
    double *b = new double[m];

    double *qa = new double [dim];
    double *qb = new double [dim];

    double *qa_gpu;
    double *qb_gpu;
    CUDA_SAFE_CALL(hipMalloc(&qa_gpu,dim*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&qb_gpu,dim*sizeof(double)));

    int i;

    b[0] = 0;
    // does nothing, just to disable valgrind warnings
    a[m-1] = 0;

    srand(time(0));

    for(i=0;i<dim;i++)
    {
        qa[i] = 0;
        qb[i] = rand()*10.0/RAND_MAX;
    }

    int incx = 1;

    double norm = 1.0/sqrt(ddot_(&dim,qb,&incx,qb,&incx));

    dscal_(&dim,&norm,qb,&incx);

    CUDA_SAFE_CALL(hipMemcpy(qa_gpu,qa,dim*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(qb_gpu,qb,dim*sizeof(double),hipMemcpyHostToDevice));

    delete [] qa;
    delete [] qb;

    norm = 1;

    double *f1 = qa_gpu;
    double *f2 = qb_gpu;
    double *tmp;

    double alpha = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for(i=1;i<m;i++)
    {
        alpha = -b[i-1];
	hipblasDscal(handle,dim,&alpha,f1,1);

	mvprod(f2,f1,norm);

	hipblasDdot(handle,dim,f1,1,f2,1,&a[i-1]);

        alpha = -a[i-1];
	hipblasDaxpy(handle,dim,&alpha,f2,1,f1,1);

	hipblasDdot(handle,dim,f1,1,f1,1,&b[i]);
	b[i] = sqrt(b[i]);

        if( fabs(b[i]) < 1e-10 )
            break;

        alpha = 1.0/b[i];

	hipblasDscal(handle,dim,&alpha,f1,1);

        tmp = f2;
        f2 = f1;
        f1 = tmp;
    }

    hipblasDestroy(handle);

    char jobz = 'N';
    int info;

    dstev_(&jobz,&m,a,&b[1],&alpha,&m,&alpha,&info);

    if(info != 0)
        std::cerr << "Error in Lanczos" << std::endl;

    alpha = a[0];

    delete [] a;
    delete [] b;

    CUDA_SAFE_CALL(hipFree(qa_gpu));
    CUDA_SAFE_CALL(hipFree(qb_gpu));

    CUDA_SAFE_CALL(hipFree(Up_data_gpu));
    CUDA_SAFE_CALL(hipFree(Up_ind_gpu));
    CUDA_SAFE_CALL(hipFree(Down_data_gpu));
    CUDA_SAFE_CALL(hipFree(Down_ind_gpu));

    CUDA_SAFE_CALL(hipFree(Umat_gpu));

    return alpha;
}


