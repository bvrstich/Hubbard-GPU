#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hipblas.h>
#include "hamgpu.h"

GPUHamiltonian::GPUHamiltonian(int Ns, int Nu, int Nd, double J, double U)
    : SparseHamiltonian(Ns,Nu,Nd,J,U)
{
}

GPUHamiltonian::~GPUHamiltonian()
{
}

__global__ void gpu_mvprod(double *x, double *y, double alpha, int NumUp, int NumDown, int dim, double *Umat, double *Down_data,unsigned int *Down_ind, int size_Down, double *Up_data, unsigned int *Up_ind, int size_Up)
{
    y[threadIdx.x] = alpha * y[threadIdx.x] + Umat[threadIdx.x] * x[threadIdx.x];

    int sv = threadIdx.x / NumDown;
    int id = threadIdx.x % NumDown;

    for(int i=0;i<size_Down;i++)
	y[threadIdx.x] += Down_data[id+i*NumDown] * x[sv*NumDown + Down_ind[id+i*NumDown]];

    for(int i=0;i<size_Up;i++)
	y[threadIdx.x] += Up_data[sv+i*NumUp] * x[id + NumDown*Up_ind[sv+i*NumUp]];
}

void GPUHamiltonian::mvprod(double *x, double *y, double alpha)
{
    int NumUp = baseUp.size();
    int NumDown = baseDown.size();
    int dim = NumUp*NumDown;

    gpu_mvprod<<<1,dim>>>(x,y,alpha,NumUp,NumDown,dim,Umat_gpu,Down_data_gpu,Down_ind_gpu,size_Down,Up_data_gpu,Up_ind_gpu,size_Up);
}

double GPUHamiltonian::LanczosDiagonalize(int m)
{
    // alloc Umat and copy to gpu
    double *Umat = Umatrix();
    hipMalloc(&Umat_gpu, getDim()*sizeof(double));
    hipMemcpy(Umat_gpu,Umat,getDim()*sizeof(double),hipMemcpyHostToDevice);

    delete [] Umat;

    int NumUp = baseUp.size();
    int NumDown = baseDown.size();

    hipMalloc(&Up_data_gpu,NumUp*size_Up*sizeof(double));
    hipMalloc(&Up_ind_gpu,NumUp*size_Up*sizeof(double));

    hipMemcpy(Up_data_gpu,Up_data,NumUp*size_Up*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(Up_ind_gpu,Up_ind,NumUp*size_Up*sizeof(double),hipMemcpyHostToDevice);

    hipMalloc(&Down_data_gpu,NumDown*size_Up*sizeof(double));
    hipMalloc(&Down_ind_gpu,NumDown*size_Up*sizeof(double));

    hipMemcpy(Down_data_gpu,Down_data,NumDown*size_Down*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(Down_ind_gpu,Down_ind,NumDown*size_Down*sizeof(double),hipMemcpyHostToDevice);

    double *a = new double[m];
    double *b = new double[m];

    double *qa = new double [dim];
    double *qb = new double [dim];

    double *qa_gpu;
    double *qb_gpu;
    hipMalloc(&qa_gpu,dim*sizeof(double));
    hipMalloc(&qb_gpu,dim*sizeof(double));

    int i;

    b[0] = 0;
    // does nothing, just to disable valgrind warnings
    a[m-1] = 0;

    srand(time(0));

    for(i=0;i<dim;i++)
    {
        qa[i] = 0;
        qb[i] = rand()*10.0/RAND_MAX;
    }

    int incx = 1;

    double norm = 1.0/sqrt(ddot_(&dim,qb,&incx,qb,&incx));

    dscal_(&dim,&norm,qb,&incx);

    hipMemcpy(qa_gpu,qa,dim*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(qb_gpu,qb,dim*sizeof(double),hipMemcpyHostToDevice);

    delete [] qa;
    delete [] qb;

    norm = 1;

    double *f1 = qa_gpu;
    double *f2 = qb_gpu;
    double *tmp;

    double alpha = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for(i=1;i<m;i++)
    {
        alpha = -b[i-1];
	hipblasDscal(handle,dim,&alpha,f1,1);

	mvprod(f2,f1,norm);

	hipblasDdot(handle,dim,f1,1,f2,1,&a[i-1]);

        alpha = -a[i-1];
	hipblasDaxpy(handle,dim,&alpha,f2,1,f1,1);

	hipblasDdot(handle,dim,f1,1,f1,1,&b[i]);
	b[i] = sqrt(b[i]);

        if( fabs(b[i]) < 1e-10 )
            break;

        alpha = 1.0/b[i];

	hipblasDscal(handle,dim,&alpha,f1,1);

        tmp = f2;
        f2 = f1;
        f1 = tmp;
    }

    hipblasDestroy(handle);

    char jobz = 'N';
    int info;

    dstev_(&jobz,&m,a,&b[1],&alpha,&m,&alpha,&info);

    if(info != 0)
        std::cerr << "Error in Lanczos" << std::endl;

    alpha = a[0];

    delete [] a;
    delete [] b;

    hipFree(qa);
    hipFree(qb);

    hipFree(Up_data_gpu);
    hipFree(Up_ind_gpu);
    hipFree(Down_data_gpu);
    hipFree(Down_ind_gpu);

    hipFree(Umat_gpu);

    return alpha;
}


