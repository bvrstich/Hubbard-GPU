#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include "hamgpu.h"

// number of threads in a block (must be multiple of 32)
#define NUMTHREADS 128

#define GRIDSIZE 65535

#define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }


GPUHamiltonian::GPUHamiltonian(int Ns, int Nu, int Nd, double J, double U)
    : SparseHamiltonian(Ns,Nu,Nd,J,U)
{
}

GPUHamiltonian::~GPUHamiltonian()
{
}

__global__ void gpu_mvprod(double *x, double *y, double alpha, int NumUp, int NumDown, int dim, double *Umat, double *Down_data,unsigned int *Down_ind, int size_Down, double *Up_data, unsigned int *Up_ind, int size_Up, int rows_shared)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if(index < dim)
    {
	double result = Umat[index] * x[index];

	int sv = index / NumDown; //__fdividef(index,NumDown);
	int id = index % NumDown; // index - sv*NumDown;

	extern __shared__ double shared[];

	unsigned int *shared_ind = (unsigned int *) &shared[size_Up * rows_shared];

	int s_sv = (blockDim.x * blockIdx.x + blockIdx.y * blockDim.x * gridDim.x)/NumDown;

	if(threadIdx.x < rows_shared && (s_sv + threadIdx.x) < NumUp)
	    for(int i=0;i<size_Up;i++)
	    {
		shared[i*rows_shared+threadIdx.x] = Up_data[s_sv + threadIdx.x + i*NumUp];

		shared_ind[i*rows_shared+threadIdx.x] = Up_ind[s_sv + threadIdx.x + i*NumUp];
	    }

	__syncthreads();

	for(int i=0;i<size_Up;i++)
	    // result += Up_data[sv+i*NumUp] * x[id + NumDown*Up_ind[sv+i*NumUp]];
	    result += shared[sv-s_sv+i*rows_shared] * x[id + NumDown*shared_ind[sv-s_sv+i*rows_shared]];

	for(int i=0;i<size_Down;i++)
	    result += Down_data[id+i*NumDown] * x[sv*NumDown + Down_ind[id+i*NumDown]];

	y[index] = alpha * y[index] + result;
    }
}

void GPUHamiltonian::mvprod(double *x, double *y, double alpha)
{
    int NumUp = baseUp.size();
    int NumDown = baseDown.size();
    int dim = NumUp*NumDown;
    dim3 numblocks(ceil(dim*1.0/NUMTHREADS));
    int rows_shared = ceil(NUMTHREADS*1.0/NumDown) + 1;
    size_t sharedmem = size_Up * rows_shared * (sizeof(double) + sizeof(unsigned int));

    if(numblocks.x > GRIDSIZE)
    {
	numblocks.x = GRIDSIZE;
	numblocks.y = ceil(ceil(dim*1.0/NUMTHREADS)*1.0/GRIDSIZE);
    }

    hipGetLastError();
    gpu_mvprod<<<numblocks,NUMTHREADS,sharedmem>>>(x,y,alpha,NumUp,NumDown,dim,Umat_gpu,Down_data_gpu,Down_ind_gpu,size_Down,Up_data_gpu,Up_ind_gpu,size_Up,rows_shared);
    CUDA_SAFE_CALL(hipGetLastError());
}

double GPUHamiltonian::LanczosDiagonalize(int m)
{
    int device;
    hipGetDevice( &device );

    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, device );

    int NumUp = baseUp.size();
    int NumDown = baseDown.size();

    size_t neededmem = getDim()*sizeof(double) +
	NumUp*size_Up*(sizeof(double)+sizeof(unsigned int)) +
	NumDown*size_Down*(sizeof(double)+sizeof(unsigned int)) +
	2*dim*sizeof(double);

    if(neededmem > prop.totalGlobalMem)
    {
	std::cerr << "Houston, we have a memory problem!" << std::endl;
	return 0;
    }

    if( ceil(dim*1.0/NUMTHREADS) > (1.0*prop.maxGridSize[0]*prop.maxGridSize[1]) ) // convert all to doubles to avoid int overflow
    {
	std::cerr << "Houston, we have a grid size problem!" << std::endl;
	return 0;
    }

    if( size_Up * (ceil(NUMTHREADS*1.0/NumDown)+1) * (sizeof(double) + sizeof(unsigned int)) > prop.sharedMemPerBlock )
    {
	std::cerr << "Houston, we have a shared memory size problem!" << std::endl;
	return 0;
    }

    // alloc Umat and copy to gpu
    double *Umat = Umatrix();
    CUDA_SAFE_CALL(hipMalloc(&Umat_gpu, dim*sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpy(Umat_gpu,Umat,dim*sizeof(double),hipMemcpyHostToDevice));

    delete [] Umat;


    CUDA_SAFE_CALL(hipMalloc(&Up_data_gpu,NumUp*size_Up*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&Up_ind_gpu,NumUp*size_Up*sizeof(unsigned int)));

    CUDA_SAFE_CALL(hipMemcpy(Up_data_gpu,Up_data,NumUp*size_Up*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(Up_ind_gpu,Up_ind,NumUp*size_Up*sizeof(unsigned int),hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMalloc(&Down_data_gpu,NumDown*size_Down*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&Down_ind_gpu,NumDown*size_Down*sizeof(unsigned int)));

    CUDA_SAFE_CALL(hipMemcpy(Down_data_gpu,Down_data,NumDown*size_Down*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(Down_ind_gpu,Down_ind,NumDown*size_Down*sizeof(unsigned int),hipMemcpyHostToDevice));

    std::vector<double> a(m,0);
    std::vector<double> b(m,0);

    double *qa = new double [dim];
    double *qb = new double [dim];

    double *qa_gpu;
    double *qb_gpu;
    CUDA_SAFE_CALL(hipMalloc(&qa_gpu,dim*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&qb_gpu,dim*sizeof(double)));

    srand(time(0));

    for(int i=0;i<dim;i++)
    {
        qa[i] = 0;
        qb[i] = (rand()*10.0/RAND_MAX);
    }

    int incx = 1;

    double norm = 1.0/sqrt(ddot_(&dim,qb,&incx,qb,&incx));

    dscal_(&dim,&norm,qb,&incx);

    CUDA_SAFE_CALL(hipMemcpy(qa_gpu,qa,dim*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(qb_gpu,qb,dim*sizeof(double),hipMemcpyHostToDevice));

    delete [] qa;
    delete [] qb;

    norm = 1;

    double *f1 = qa_gpu;
    double *f2 = qb_gpu;
    double *tmp;

    double alpha = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

//    hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_DEVICE;
//    hipblasSetPointerMode(handle,mode);

    int i=1;

    std::vector<double> acopy(a);
    std::vector<double> bcopy(b);

    double E = 1;

    hipEvent_t start, stop;
    float exeTime;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    hipEventRecord(start, 0);

    while(fabs(E-acopy[0]) > 1e-4)
    {
	E = acopy[0];

	for(;i<m;i++)
	{
	    alpha = -b[i-1];
	    hipblasDscal(handle,dim,&alpha,f1,1);

	    mvprod(f2,f1,norm);

	    hipblasDdot(handle,dim,f1,1,f2,1,&a[i-1]);

	    alpha = -a[i-1];
	    hipblasDaxpy(handle,dim,&alpha,f2,1,f1,1);

	    hipblasDdot(handle,dim,f1,1,f1,1,&b[i]);
	    b[i] = sqrt(b[i]);

	    if( fabs(b[i]) < 1e-10 )
		break;

	    alpha = 1.0/b[i];

	    hipblasDscal(handle,dim,&alpha,f1,1);

	    tmp = f2;
	    f2 = f1;
	    f1 = tmp;
	}

	acopy = a;
	bcopy = b;

	char jobz = 'N';
	int info;

	dstev_(&jobz,&m,acopy.data(),&bcopy.data()[1],&alpha,&m,&alpha,&info);

	if(info != 0)
	    std::cerr << "Error in Lanczos" << std::endl;

	m += 10;
	a.resize(m);
	b.resize(m);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime( &exeTime, start, stop );

    std::cout << "Done in " << m-10 << " Iterations" << std::endl;
    std::cout << "Cuda time: " << exeTime/1000 << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(handle);

    alpha = acopy[0];

    CUDA_SAFE_CALL(hipFree(qa_gpu));
    CUDA_SAFE_CALL(hipFree(qb_gpu));

    CUDA_SAFE_CALL(hipFree(Up_data_gpu));
    CUDA_SAFE_CALL(hipFree(Up_ind_gpu));
    CUDA_SAFE_CALL(hipFree(Down_data_gpu));
    CUDA_SAFE_CALL(hipFree(Down_ind_gpu));

    CUDA_SAFE_CALL(hipFree(Umat_gpu));

    CUDA_SAFE_CALL(hipDeviceReset());

    return alpha;
}

/* vim: set ts=8 sw=4 tw=0 expandtab :*/
