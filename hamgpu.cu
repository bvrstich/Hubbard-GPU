#include "hip/hip_runtime.h"
/* Copyright (C) 2012  Ward Poelmans

This file is part of Hubbard-GPU.

Hubbard-GPU is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

Hubbard-GPU is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Hubbard-GPU.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include "hamgpu.h"
#include "hamsparse.h"
#include "hamsparse2D.h"

// number of threads in a block (must be multiple of 32)
#define NUMTHREADS 128

// the maximum size of the grid
#define GRIDSIZE 65535

// Helper macro to check CUDA return values
#define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }


/**
  * The constructor for the SparseHamiltonian Class
  */
template<>
GPUHamiltonian<SparseHamiltonian>::GPUHamiltonian(int Ns, int Nu, int Nd, double J, double U)
    : SparseHamiltonian(Ns,Nu,Nd,J,U)
{
}

/**
  * The constructor for the SparseHamiltonian2D Class
  */
template<>
GPUHamiltonian<SparseHamiltonian2D>::GPUHamiltonian(int L, int D, int Nu, int Nd, double J, double U)
    : SparseHamiltonian2D(L,D,Nu,Nd,J,U)
{
}

template<class T>
GPUHamiltonian<T>::~GPUHamiltonian()
{
}

/**
  * The actual Cuda kernel to calculate the matrix vector product with the hamiltonian
  */
__global__ void gpu_mvprod(double *x, double *y, double alpha, int NumUp, int NumDown, int dim, double *Umat, double *Down_data,unsigned int *Down_ind, int size_Down, double *Up_data, unsigned int *Up_ind, int size_Up, int rows_shared)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if(index < dim)
    {
	double result = Umat[index] * x[index];

	int sv = index / NumDown; //__fdividef(index,NumDown);
	int id = index % NumDown; // index - sv*NumDown;

	extern __shared__ double shared[];

	unsigned int *shared_ind = (unsigned int *) &shared[size_Up * rows_shared];

	int s_sv = (blockDim.x * blockIdx.x + blockIdx.y * blockDim.x * gridDim.x)/NumDown;

	if(threadIdx.x < rows_shared && (s_sv + threadIdx.x) < NumUp)
	    for(int i=0;i<size_Up;i++)
	    {
		shared[i*rows_shared+threadIdx.x] = Up_data[s_sv + threadIdx.x + i*NumUp];

		shared_ind[i*rows_shared+threadIdx.x] = Up_ind[s_sv + threadIdx.x + i*NumUp];
	    }

	__syncthreads();

	for(int i=0;i<size_Up;i++)
	    // result += Up_data[sv+i*NumUp] * x[id + NumDown*Up_ind[sv+i*NumUp]];
	    result += shared[sv-s_sv+i*rows_shared] * x[id + NumDown*shared_ind[sv-s_sv+i*rows_shared]];

	for(int i=0;i<size_Down;i++)
	    result += Down_data[id+i*NumDown] * x[sv*NumDown + Down_ind[id+i*NumDown]];

	y[index] = alpha * y[index] + result;
    }
}

/**
 * The matrix vector product. The method should calculate y = A*x + alpha * y
 * @param x the input vector
 * @param y the output vector
 * @param alpha the multiplicative constant
 */
template<class T>
void GPUHamiltonian<T>::mvprod(double *x, double *y, double alpha) const
{
    int NumUp = T::baseUp.size();
    int NumDown = T::baseDown.size();
    int dim = NumUp*NumDown;
    dim3 numblocks(ceil(dim*1.0/NUMTHREADS));
    int rows_shared = ceil(NUMTHREADS*1.0/NumDown) + 1;
    size_t sharedmem = T::size_Up * rows_shared * (sizeof(double) + sizeof(unsigned int));

    if(numblocks.x > GRIDSIZE)
    {
	numblocks.x = GRIDSIZE;
	numblocks.y = ceil(ceil(dim*1.0/NUMTHREADS)*1.0/GRIDSIZE);
    }

    hipGetLastError();
    gpu_mvprod<<<numblocks,NUMTHREADS,sharedmem>>>(x,y,alpha,NumUp,NumDown,dim,Umat_gpu,Down_data_gpu,Down_ind_gpu,T::size_Down,Up_data_gpu,Up_ind_gpu,T::size_Up,rows_shared);
    CUDA_SAFE_CALL(hipGetLastError());
}

/**
 * Calculates the lowest eigenvalue of the hamiltonian matrix using
 * the lanczos algorithm. Needs lapack.
 * @param m an optional estimate for the lanczos space size
 * @return the lowest eigenvalue
 */
template<class T>
double GPUHamiltonian<T>::LanczosDiagonalize(int m)
{
    if(!m)
        m = 10;

    int device;
    hipGetDevice( &device );

    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, device );

    int NumUp = T::baseUp.size();
    int NumDown = T::baseDown.size();

    size_t neededmem = T::getDim()*sizeof(double) +
	NumUp*T::size_Up*(sizeof(double)+sizeof(unsigned int)) +
	NumDown*T::size_Down*(sizeof(double)+sizeof(unsigned int)) +
	2*T::dim*sizeof(double);

    if(neededmem > prop.totalGlobalMem)
    {
	std::cerr << "Houston, we have a memory problem!" << std::endl;
	return 0;
    }

    if( ceil(T::dim*1.0/NUMTHREADS) > (1.0*prop.maxGridSize[0]*prop.maxGridSize[1]) ) // convert all to doubles to avoid int overflow
    {
	std::cerr << "Houston, we have a grid size problem!" << std::endl;
	return 0;
    }

    if( T::size_Up * (ceil(NUMTHREADS*1.0/NumDown)+1) * (sizeof(double) + sizeof(unsigned int)) > prop.sharedMemPerBlock )
    {
	std::cerr << "Houston, we have a shared memory size problem!" << std::endl;
	return 0;
    }

    // alloc Umat and copy to gpu
    double *Umat = T::Umatrix();
    CUDA_SAFE_CALL(hipMalloc(&Umat_gpu, T::dim*sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpy(Umat_gpu,Umat,T::dim*sizeof(double),hipMemcpyHostToDevice));

    delete [] Umat;


    CUDA_SAFE_CALL(hipMalloc(&Up_data_gpu,NumUp*T::size_Up*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&Up_ind_gpu,NumUp*T::size_Up*sizeof(unsigned int)));

    CUDA_SAFE_CALL(hipMemcpy(Up_data_gpu,T::Up_data,NumUp*T::size_Up*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(Up_ind_gpu,T::Up_ind,NumUp*T::size_Up*sizeof(unsigned int),hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMalloc(&Down_data_gpu,NumDown*T::size_Down*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&Down_ind_gpu,NumDown*T::size_Down*sizeof(unsigned int)));

    CUDA_SAFE_CALL(hipMemcpy(Down_data_gpu,T::Down_data,NumDown*T::size_Down*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(Down_ind_gpu,T::Down_ind,NumDown*T::size_Down*sizeof(unsigned int),hipMemcpyHostToDevice));

    std::vector<double> a(m,0);
    std::vector<double> b(m,0);

    double *qa = new double [T::dim];
    double *qb = new double [T::dim];

    double *qa_gpu;
    double *qb_gpu;
    CUDA_SAFE_CALL(hipMalloc(&qa_gpu,T::dim*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&qb_gpu,T::dim*sizeof(double)));

    srand(time(0));

    for(int i=0;i<T::dim;i++)
    {
        qa[i] = 0;
        qb[i] = (rand()*10.0/RAND_MAX);
    }

    int incx = 1;
    int dim = T::dim;

    double norm = 1.0/sqrt(ddot_(&dim,qb,&incx,qb,&incx));

    dscal_(&dim,&norm,qb,&incx);

    CUDA_SAFE_CALL(hipMemcpy(qa_gpu,qa,T::dim*sizeof(double),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(qb_gpu,qb,T::dim*sizeof(double),hipMemcpyHostToDevice));

    delete [] qa;
    delete [] qb;

    norm = 1;

    double *f1 = qa_gpu;
    double *f2 = qb_gpu;
    double *tmp;

    double alpha = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

//    hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_DEVICE;
//    hipblasSetPointerMode(handle,mode);

    int i=1;

    std::vector<double> acopy(a);
    std::vector<double> bcopy(b);

    double E = 1;

    hipEvent_t start, stop;
    float exeTime;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    hipEventRecord(start, 0);

    while(fabs(E-acopy[0]) > 1e-4)
    {
	E = acopy[0];

	for(;i<m;i++)
	{
	    alpha = -b[i-1];
	    hipblasDscal(handle,T::dim,&alpha,f1,1);

	    mvprod(f2,f1,norm);

	    hipblasDdot(handle,T::dim,f1,1,f2,1,&a[i-1]);

	    alpha = -a[i-1];
	    hipblasDaxpy(handle,T::dim,&alpha,f2,1,f1,1);

	    hipblasDdot(handle,T::dim,f1,1,f1,1,&b[i]);
	    b[i] = sqrt(b[i]);

	    if( fabs(b[i]) < 1e-10 )
		break;

	    alpha = 1.0/b[i];

	    hipblasDscal(handle,T::dim,&alpha,f1,1);

	    tmp = f2;
	    f2 = f1;
	    f1 = tmp;
	}

	acopy = a;
	bcopy = b;

	char jobz = 'N';
	int info;

	dstev_(&jobz,&m,acopy.data(),&bcopy.data()[1],&alpha,&m,&alpha,&info);

	if(info != 0)
	    std::cerr << "Error in Lanczos" << std::endl;

	m += 10;
	a.resize(m);
	b.resize(m);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime( &exeTime, start, stop );

    std::cout << "Done in " << m-10 << " Iterations" << std::endl;
    std::cout << "Cuda time: " << exeTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(handle);

    alpha = acopy[0];

    CUDA_SAFE_CALL(hipFree(qa_gpu));
    CUDA_SAFE_CALL(hipFree(qb_gpu));

    CUDA_SAFE_CALL(hipFree(Up_data_gpu));
    CUDA_SAFE_CALL(hipFree(Up_ind_gpu));
    CUDA_SAFE_CALL(hipFree(Down_data_gpu));
    CUDA_SAFE_CALL(hipFree(Down_ind_gpu));

    CUDA_SAFE_CALL(hipFree(Umat_gpu));

    CUDA_SAFE_CALL(hipDeviceReset());

    return alpha;
}

// Expliciet specify the template class with the possible template parameters
template class GPUHamiltonian<SparseHamiltonian>;
template class GPUHamiltonian<SparseHamiltonian2D>;

/* vim: set ts=8 sw=4 tw=0 expandtab :*/
